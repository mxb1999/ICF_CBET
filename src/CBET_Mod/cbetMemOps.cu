#include "CBET_Interface.hpp"
#include "parallelConfig.hpp"

void cbetOptimize()
{
    if(cudaCalc)
    {
        hipMemAdvise(machnum, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
        hipMemAdvise(u_flow, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);

        hipMemAdvise(dkx, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
        hipMemAdvise(dkz, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
        hipMemAdvise(dkmag, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);

        hipMemAdvise(i_b_new, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
        hipMemAdvise(i_b, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);


        hipMemPrefetchAsync(machnum, sizeof(double)*GRID, 0);
        hipMemPrefetchAsync(u_flow, sizeof(double)*GRID, 0);

        hipMemPrefetchAsync(dkx, sizeof(double)*CROSS, 0);
        hipMemPrefetchAsync(dkz, sizeof(double)*CROSS, 0);
        hipMemPrefetchAsync(dkmag, sizeof(double)*CROSS, 0);

        hipMemPrefetchAsync(i_b_new, sizeof(double)*CROSS, 0);
        hipMemPrefetchAsync(i_b, sizeof(double)*CROSS, 0);
    }
}


void freeCBETArrs()
{
  if(cudaCalc)
  {
    hipFree(i_b_new);
    hipFree(i_b);
    hipFree(machnum);
    hipFree(u_flow);
    hipFree(dkx);
    hipFree(dkz);
    hipFree(dkmag);
  }else
  {
    delete [] i_b_new;
    delete [] i_b;
    delete [] machnum;
    delete [] u_flow;
    delete [] dkx;
    delete [] dkz;
    delete [] dkmag;
  }
}