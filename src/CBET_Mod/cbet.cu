#include "hip/hip_runtime.h"
#include "CBET_Interface.hpp"
#include "parallelConfig.hpp"


//define CBET Gain Function
__global__ void 
cbetGain(CBETVars* constants, CBETArrs* arrays)
{

    int nbeams_cu = constants->nbeams_cu;
    int nrays_cu = constants->nrays_cu;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int beam = index / nrays_cu;
    if(beam >= (nbeams_cu))
    {
        return;
    }

    int raynum = index % nrays_cu;
    //imported constants
    int ncrossings_cu = constants->ncrossings_cu;
    int nx_cu = constants->nx_cu;
    int nz_cu = constants->nz_cu;
    int numstored_cu = constants->numstored_cu;
    double dx_cu = constants->dx_cu;
    double dz_cu = constants->dz_cu;
    double ncrit_cu = constants->ncrit_cu;
    double c_cu = constants->c_cu;
    double pi_cu = constants->pi_cu;
    double iaw_cu = constants->iaw_cu;
    double cs_cu = constants->cs_cu;
    double estat_cu = constants->estat_cu;
    double Ti_cu = constants->Ti_cu;
    double Te_cu = constants->Te_cu;
    double Z_cu = constants->Z_cu;
    double omega_cu = constants->omega_cu;
    double kb_cu = constants->kb_cu;
    double me_cu = constants->me_cu;

    //imported arrays
    double* i_b_cu = arrays->i_b_cu;
    double* i_b_new_cu = arrays->i_b_new_cu;

    double* W_cu = arrays->W_cu;

    double* x_cu = arrays->x_cu;

    double* z_cu = arrays->z_cu;
    double* W_new_cu = arrays->W_new_cu;
    double* dkx_cu = arrays->dkx_cu;
    double* dkz_cu = arrays->dkz_cu;
    double* dkmag_cu = arrays->dkmag_cu;
    double* uflow_cu = arrays->uflow_cu;

    int* ints_cu = arrays->ints_cu;
    double* eden_cu = arrays->eden_cu;
    int* boxes_cu = arrays->boxes_cu;
    int* numrays_cu = arrays->numrays_cu;
    int* present_cu = arrays->present_cu;    
    //iterate over each ray beam (excepting the last one)
    //each beam will be treated as a pump beam for those preceeding, as a seed beam for those following
    double constant1 = (pow(estat_cu,2.0))/(4*(1.0e3*me_cu)*c_cu*omega_cu*kb_cu*Te_cu*(1+3*Ti_cu/(Z_cu*Te_cu)));
    for(int m = 0; m < ncrossings_cu; m++)
    {
        int ix = vec4D_cu(boxes_cu, beam,raynum,m,0, nrays_cu, ncrossings_cu, 2);
        int iz = vec4D_cu(boxes_cu, beam,raynum,m,1, nrays_cu, ncrossings_cu, 2);
        if(!ix || !iz)
        {
            break;
        }
        if(!vec4D_cu(ints_cu, beam, raynum, m, 0, nrays_cu, ncrossings_cu, numstored_cu))
        {
            continue;
        }

        ix--;
        iz--;
        int index = 0;
        //find all rays that interact with raynum in the other beam(s) of higher order
        for(int q = 0; q < nbeams_cu; q++)
        {
            if(q == beam)
            {
                continue;
            }
            //find the number of rays interacted with
            int cnt = vec4D_cu(numrays_cu, beam,raynum,m,q, nrays_cu, ncrossings_cu, nbeams_cu);
            int* crossInd = new int[cnt]{0};

            for(int l = index; l < index+cnt; l++)//get the crossing index for all rays of beam q in (ix,iz)
            {
                for(int p = 0; p < ncrossings_cu;p++)
                {
                    
                    int ox = vec4D_cu(boxes_cu, q,l,p,0, nrays_cu, ncrossings_cu, 2)-1;
                    int oz = vec4D_cu(boxes_cu, q,l,p,1, nrays_cu, ncrossings_cu, 2)-1;
                    if(!ox || !oz)
                    {
                        break;
                    }
                    
                    if(ox == ix && oz == iz)
                    {
                        crossInd[l] = p;
                        break;
                    }
                    
                }
            }
            //double xprev = x_cu[ix];
            //double zprev = z_cu[ix];
            int nlim = (vec3D_cu(present_cu,beam,ix,iz, nx_cu, nz_cu) > cnt) ? cnt : vec3D_cu(present_cu,beam,ix,iz, nx_cu, nz_cu);
            for(int n = 0; n < nlim; n++)
            {
                
                int rayOther = vec4D_cu(ints_cu, beam, raynum, m, index + n, nrays_cu, ncrossings_cu, numstored_cu);
                int rayCross = crossInd[n];
                double mag1 = vec3D_cu(dkmag_cu, beam, raynum, m, nrays_cu, ncrossings_cu);
                double mag2 = vec3D_cu(dkmag_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu);
                if(mag2 < 1.0*dx_cu)
                {
                    continue;
                }
                double ne = vec2D_cu(eden_cu, ix,iz,nz_cu);
                double epsilon = 1.0-ne/ncrit_cu;
                double kmag = (omega_cu/c_cu)*sqrt(epsilon);

                double kx1 = kmag * vec3D_cu(dkx_cu, beam, raynum, m, nrays_cu, ncrossings_cu) / mag1;
                double kx2 = kmag * vec3D_cu(dkx_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu) / mag2;

                double kz1 = kmag * vec3D_cu(dkz_cu, beam, raynum, m, nrays_cu, ncrossings_cu) / mag1;
                double kz2 = kmag * vec3D_cu(dkz_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu) / mag2;
                double kiaw = sqrt(pow(kx2-kx1,2.0)+pow(kz2-kz1,2.0));

                double ws = kiaw*cs_cu;
                double omega1 = omega_cu;
                double omega2 = omega_cu;
                double eta = ((omega2-omega1)-(kx2-kx1)*vec2D_cu(uflow_cu,ix,iz,nz_cu))/(ws+1.0e-10);
                double efield2 = sqrt(8.*pi_cu*1.0e7*vec3D_cu(i_b_new_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu)/c_cu);   
                double P = (pow(iaw_cu,2)*eta)/(pow((pow(eta,2)-1.0),2)+pow((iaw_cu),2)*pow(eta,2));  
                double gain1 = constant1*pow(efield2,2)*(ne/ncrit_cu)*(1/iaw_cu)*P;               //L^-1 from Russ's paper
                double oldEnergy1 = vec3D_cu(W_new_cu, beam,raynum,m,nrays_cu, ncrossings_cu);
                double oldEnergy2 = ((beam < q) ? 1 : -1)*vec3D_cu(W_new_cu, q,rayOther,rayCross,nrays_cu, ncrossings_cu);
                double newEnergy1Mult = exp(oldEnergy2*mag1*gain1/sqrt(epsilon));
                vec3DW_cu(W_cu, beam, raynum, m, nrays_cu, ncrossings_cu,oldEnergy1);
                vec3DM_cu(W_new_cu, beam, raynum, m, nrays_cu, ncrossings_cu,newEnergy1Mult);
            }
        }
    }

    
}
__device__ void
elimCrossEffect(int currcross, int beamOther, int rayOther, int rayCross, CBETVars* constants, CBETArrs* arrays)
{
    int nbeams_cu = constants->nbeams_cu;
    int nrays_cu = constants->nrays_cu;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int beam = index / nrays_cu;
    if(beam >= (nbeams_cu - 1))
    {
        return;
    }
    printf("Rectifying\n");
    int raynum = index % nrays_cu;
    //imported constants
    int ncrossings_cu = constants->ncrossings_cu;
    int nx_cu = constants->nx_cu;
    int nz_cu = constants->nz_cu;
    int numstored_cu = constants->numstored_cu;
    double dx_cu = constants->dx_cu;
    double dz_cu = constants->dz_cu;
    double ncrit_cu = constants->ncrit_cu;
    double c_cu = constants->c_cu;
    double pi_cu = constants->pi_cu;
    double iaw_cu = constants->iaw_cu;
    double cs_cu = constants->cs_cu;
    double estat_cu = constants->estat_cu;
    double Ti_cu = constants->Ti_cu;
    double Te_cu = constants->Te_cu;
    double Z_cu = constants->Z_cu;
    double omega_cu = constants->omega_cu;
    double kb_cu = constants->kb_cu;
    double me_cu = constants->me_cu;
    double constant1 = (pow(estat_cu,2.0))/(4*(1.0e3*me_cu)*c_cu*omega_cu*kb_cu*Te_cu*(1+3*Ti_cu/(Z_cu*Te_cu)));

    //imported arrays
    double* i_b_cu = arrays->i_b_cu;
    double* i_b_new_cu = arrays->i_b_new_cu;

    double* W_cu = arrays->W_cu;

    double* x_cu = arrays->x_cu;

    double* z_cu = arrays->z_cu;
    double* W_new_cu = arrays->W_new_cu;
    double* dkx_cu = arrays->dkx_cu;
    double* dkz_cu = arrays->dkz_cu;
    double* dkmag_cu = arrays->dkmag_cu;
    double* uflow_cu = arrays->uflow_cu;
    int m = currcross;
    int q = beamOther;
    int* ints_cu = arrays->ints_cu;
    double* eden_cu = arrays->eden_cu;
    int* boxes_cu = arrays->boxes_cu;
    int* numrays_cu = arrays->numrays_cu;
    int* present_cu = arrays->present_cu;    
    int ix = vec4D_cu(boxes_cu, beam,raynum,m,0, nrays_cu, ncrossings_cu, 2)-1;
    int iz = vec4D_cu(boxes_cu, beam,raynum,m,1, nrays_cu, ncrossings_cu, 2)-1;
    double mag1 = vec3D_cu(dkmag_cu, beam, raynum, m, nrays_cu, ncrossings_cu);
    double mag2 = vec3D_cu(dkmag_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu);
    double ne = vec2D_cu(eden_cu, ix,iz,nz_cu);
    double epsilon = 1.0-ne/ncrit_cu;
    double kmag = (omega_cu/c_cu)*sqrt(epsilon);

    double kx1 = kmag * vec3D_cu(dkx_cu, beam, raynum, m, nrays_cu, ncrossings_cu) / mag1;
    double kx2 = kmag * vec3D_cu(dkx_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu) / mag2;

    double kz1 = kmag * vec3D_cu(dkz_cu, beam, raynum, m, nrays_cu, ncrossings_cu) / mag1;
    double kz2 = kmag * vec3D_cu(dkz_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu) / mag2;
    double kiaw = sqrt(pow(kx2-kx1,2.0)+pow(kz2-kz1,2.0));

    double ws = kiaw*cs_cu;
    double omega1 = omega_cu;
    double omega2 = omega_cu;
    double eta = ((omega2-omega1)-(kx2-kx1)*vec2D_cu(uflow_cu,ix,iz,nz_cu))/(ws+1.0e-10);
    double efield2 = sqrt(8.*pi_cu*1.0e7*vec3D_cu(i_b_new_cu, q, rayOther, rayCross, nrays_cu, ncrossings_cu)/c_cu);   
    double P = (pow(iaw_cu,2)*eta)/(pow((pow(eta,2)-1.0),2)+pow((iaw_cu),2)*pow(eta,2));  
    double gain1 = constant1*pow(efield2,2)*(ne/ncrit_cu)*(1/iaw_cu)*P;               //L^-1 from Russ's paper
    double oldEnergy2 = ((beam < q) ? 1 : -1)*vec3D_cu(W_new_cu, q,rayOther,rayCross,nrays_cu, ncrossings_cu);
    double energyCorrection = exp(-1*oldEnergy2*mag1*gain1/sqrt(epsilon));
    vec3DM_cu(W_new_cu, beam, raynum, m, nrays_cu, ncrossings_cu,energyCorrection);
}
__global__ void 
cbetUpdate(CBETVars* constants, CBETArrs* arrays, int* cbetKill, int beam)
{
    
    int nbeams_cu = constants->nbeams_cu;
    int nrays_cu = constants->nrays_cu;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index >= (nrays_cu))
    {
        return;
    }

    int raynum = index % nrays_cu;
    //imported constants
    int ncrossings_cu = constants->ncrossings_cu;
    int nx_cu = constants->nx_cu;
    int nz_cu = constants->nz_cu;
    int numstored_cu = constants->numstored_cu;
    double dx_cu = constants->dx_cu;
    double dz_cu = constants->dz_cu;
    double ncrit_cu = constants->ncrit_cu;
    double c_cu = constants->c_cu;
    double pi_cu = constants->pi_cu;
    double iaw_cu = constants->iaw_cu;
    double cs_cu = constants->cs_cu;
    double estat_cu = constants->estat_cu;
    double Ti_cu = constants->Ti_cu;
    double Te_cu = constants->Te_cu;
    double Z_cu = constants->Z_cu;
    double omega_cu = constants->omega_cu;
    double kb_cu = constants->kb_cu;
    double me_cu = constants->me_cu;

    //imported arrays
    double* i_b_cu = arrays->i_b_cu;
    double* i_b_new_cu = arrays->i_b_new_cu;

    double* W_cu = arrays->W_cu;

    double* x_cu = arrays->x_cu;

    double* z_cu = arrays->z_cu;
    double* W_new_cu = arrays->W_new_cu;
    double* dkx_cu = arrays->dkx_cu;
    double* dkz_cu = arrays->dkz_cu;
    double* dkmag_cu = arrays->dkmag_cu;
    double* uflow_cu = arrays->uflow_cu;
    printf("Rectifying\n");

    int* ints_cu = arrays->ints_cu;
    double* eden_cu = arrays->eden_cu;
    int* boxes_cu = arrays->boxes_cu;
    int* numrays_cu = arrays->numrays_cu;
    int* present_cu = arrays->present_cu;    
    //iterate over each ray beam (excepting the last one)
    //each beam will be treated as a pump beam for those preceeding, as a seed beam for those following
    double constant1 = (pow(estat_cu,2.0))/(4*(1.0e3*me_cu)*c_cu*omega_cu*kb_cu*Te_cu*(1+3*Ti_cu/(Z_cu*Te_cu)));
    int contact = 0;
    for(int m = 0; m < ncrossings_cu; m++)
    {
        int ix = vec4D_cu(boxes_cu, beam,raynum,m,0, nrays_cu, ncrossings_cu, 2);
        int iz = vec4D_cu(boxes_cu, beam,raynum,m,1, nrays_cu, ncrossings_cu, 2);
        if(!ix || !iz)
        {
            break;
        }

        if(!vec4D_cu(ints_cu, beam, raynum, m, 0, nrays_cu, ncrossings_cu, numstored_cu))
        {
            continue;
        }
        if(!contact)
        {
            contact = m;
        }
        ix--;
        iz--;
        int index = 0;
        //find all rays that interact with raynum in the other beam(s) of higher order
        for(int q = 0; q < nbeams_cu; q++)
        {
            if(q == beam)
            {
                continue;
            }
            //find the number of rays interacted with
            int cnt = vec4D_cu(numrays_cu, beam,raynum,m,q, nrays_cu, ncrossings_cu, nbeams_cu);
            int* crossInd = new int[cnt]{0};
            for(int l = index; l < cnt; l++)//get the crossing index for all rays of beam q in (ix,iz)
            {
                for(int p = 0; p < ncrossings_cu;p++)
                {
                    int ox = vec4D_cu(boxes_cu, q,l,p,0, nrays_cu, ncrossings_cu, 2)-1;
                    int oz = vec4D_cu(boxes_cu, q,l,p,1, nrays_cu, ncrossings_cu, 2)-1;
                    if(ox == ix && oz == iz)
                    {
                        crossInd[l] = p;
                        break;
                    }
                }
                int killed = vec2D_cu(cbetKill, q,l,nrays_cu);
                if(killed && ((killed-1) < crossInd[l]))
                {
                    elimCrossEffect(m, q,l,crossInd[l], constants, arrays);
                }
            }
            double prevIntensitySeed = vec3D_cu(i_b_new_cu, beam,raynum,m,nrays_cu, ncrossings_cu);
            double intensityMultSeed = (1.0 - (vec3D_cu(W_new_cu, beam,raynum,m,nrays_cu, ncrossings_cu)/vec3D_cu(W_cu, beam,raynum,m,nrays_cu, ncrossings_cu)));
            double dI1 = (-1.0*intensityMultSeed*prevIntensitySeed);
            int kill = 0;
            int transfer = 0;
            if(abs(dI1) > prevIntensitySeed && dI1 < 0)
            {
                kill = 1;
                printf("killed at: %d\n", m-contact);
            }
            if(prevIntensitySeed <= 0)
            {
                kill = 1;
                prevIntensitySeed = 0;
            }
            //printf("%e\n", vec3D_cu(W_new_cu, beam,raynum,m,nrays_cu, ncrossings_cu)/vec3D_cu(W_cu, beam,raynum,m,nrays_cu, ncrossings_cu));

            for(int l = m; l < ncrossings_cu; l++)
            {
                double newVal;
                if(!kill)
                {
                    vec3DI_cu(i_b_new_cu, beam,raynum,l,nrays_cu, ncrossings_cu,dI1);
                    newVal = prevIntensitySeed+dI1;
                }else
                {
                    vec3DW_cu(i_b_new_cu, beam,raynum,l,nrays_cu, ncrossings_cu,0.0);
                    newVal = 0.0;
                }
            }
            if(kill)
            {
                vec2DW_cu(cbetKill, beam, raynum, nrays_cu, m+1);
            }
            
            
        }
    }

    
}

void launchCBETKernel()
{
    printf("CBET\n");
    initArrays();
    CBETVars* vars = new_cbetVars();
    CBETArrs* arrays = new_cbetArrs();
    int* cbetKill;
    hipMallocManaged(&cbetKill, sizeof(int)*RAYS);
    for(int i = 0; i < RAYS; i++)
    {
        cbetKill[i] = 0;
    }
    int B1 = nrays*nbeams/256+1;
    int B2 = nrays/256+1;

    for(int i = 0; i < 1; i++)
    {
        cbetGain<<<B1, 256>>>(vars, arrays);
        hipDeviceSynchronize();
        for(int i = 0; i < nbeams;i++)
        {
            cbetUpdate<<<B2, 256>>>(vars, arrays,cbetKill, i);
            hipDeviceSynchronize();
        }
        
    }

    printf("%e\n", cs);
}

