#include "GPU/cuda_help"


//GPU IO handlers
int CGPUMemCpy(gconfig* gpu, void* dest, void* source, size_t size, int direction)
{
    if(gpu->type != CUDA)
    {
        printf("Non-CUDA GPU called CUDA function\n");
        return 0;
    }
    int dir = (direction) ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost;
    hipMemcpy(dest, source, size, dir);
    return 1;
    //Allocate host memory

};//allocate memory on gpui
int CGPUAlloc(gconfig* gpu, void* p, size_t size)
{
    if(gpu->type != CUDA)
    {
        printf("Non-CUDA GPU called CUDA function\n");
        return 0;
    }
    hipMalloc(&p, size);
    return 1;
};//allocate memory on gpu in pointer p

