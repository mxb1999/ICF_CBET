#include "CBET_Interface.hpp"
#include "parallelConfig.hpp"
#include "cuda_help.hpp"

void cbetOptimize()
{
    if(cudaCalc)
    {
        hipMemAdvise(machnum, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
        hipMemAdvise(u_flow, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);

        hipMemAdvise(dk, sizeof(double)*CROSS*3, hipMemAdviseSetReadMostly, 0);
        hipMemAdvise(dkmag, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);

        hipMemAdvise(i_b_new, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
        hipMemAdvise(i_b, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);


        hipMemPrefetchAsync(machnum, sizeof(double)*GRID, 0);
        hipMemPrefetchAsync(u_flow, sizeof(double)*GRID, 0);

        hipMemPrefetchAsync(dk, sizeof(double)*CROSS*3, 0);
        hipMemPrefetchAsync(dkmag, sizeof(double)*CROSS, 0);

        hipMemPrefetchAsync(i_b_new, sizeof(double)*CROSS, 0);
        hipMemPrefetchAsync(i_b, sizeof(double)*CROSS, 0);
    }
}


void freeCBETArrs()
{

  if(cudaCalc)
  {
    hipError_t err = hipFree(i_b_new);
    err = hipFree(i_b);
    err = hipFree(machnum);
    err = hipFree(u_flow);
    err = hipFree(dk);
    err = hipFree(dkmag);
  }else
  {
    delete [] i_b_new;
    delete [] i_b;
    delete [] machnum;
    delete [] u_flow;
    delete [] dk;
    delete [] dkmag;
  }

}