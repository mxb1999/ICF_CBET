#include <stdio.h>
#include <stdlib.h>
#include "Trace_interface.hpp"
#include "io_interface.hpp"
#include "cuda_help.hpp"
#include <stdarg.h>

void optimizePreTraceArrs()
{
  if(cudaCalc)
  {
    hipMemAdvise(dedendx, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(dedendz, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(x, sizeof(double)*nx, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(z, sizeof(double)*nz, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(eden, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(marked, sizeof(int)*GRID*numstored*nbeams, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(boxes, sizeof(int)*CROSS*2, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(present, sizeof(int)*GRID*nbeams, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(crossesx, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(crossesz, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
  }
}
void optimizePostTraceArrs()
{
  if(cudaCalc)
  {
    hipMemAdvise(marked, sizeof(int)*GRID*numstored*nbeams, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(boxes, sizeof(int)*CROSS*2, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(present, sizeof(int)*GRID*nbeams, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(crossesx, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(crossesz, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
  }
}

void freeTraceArrs()
{
  if(cudaCalc)
  {
    //hipError_t err = hipFree((void*)dedendx);
    //printf("1 %s\n", hipGetErrorString(err));
    //err = hipFree(dedendz);
    //printf("2 %s\n", hipGetErrorString(err));

    hipError_t err = hipFree(x);
    err = hipFree(z);
    err = hipFree(eden);
    err = hipFree(marked);
    err = hipFree(present);

    err = hipFree(boxes);

    err = hipFree(wpe);

    //err = hipFree(crossesx);
    //printf("10 %s\n", hipGetErrorString(err));
    //err = hipFree(crossesz);
    //printf("11 %s\n", hipGetErrorString(err));
  }else
  {
    delete [] dedendx;
    delete [] dedendz;
    delete [] x;
    delete [] z;
    delete [] eden;
    delete [] marked;
    delete [] present;
    delete [] boxes;
    delete [] wpe;
    delete [] crossesx;
    delete [] crossesz;
  }
}