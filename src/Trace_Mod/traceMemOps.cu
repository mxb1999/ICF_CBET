#include <stdio.h>
#include <stdlib.h>
#include "Trace_interface.hpp"
#include "io_interface.hpp"
#include "cuda_help.hpp"
#include <stdarg.h>

void optimizePreTraceArrs()
{
  if(cudaCalc)
  {
    hipMemAdvise(dedendx, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(dedendz, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(x, sizeof(double)*nx, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(z, sizeof(double)*nz, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(eden, sizeof(double)*GRID, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(marked, sizeof(int)*GRID*numstored*nbeams, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(boxes, sizeof(int)*CROSS*2, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(present, sizeof(int)*GRID*nbeams, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(crossesx, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(crossesz, sizeof(double)*CROSS, hipMemAdviseSetPreferredLocation, 0);
  }
}
void optimizePostTraceArrs()
{
  if(cudaCalc)
  {
    hipMemAdvise(marked, sizeof(int)*GRID*numstored*nbeams, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(boxes, sizeof(int)*CROSS*2, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(present, sizeof(int)*GRID*nbeams, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(crossesx, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(crossesz, sizeof(double)*CROSS, hipMemAdviseSetReadMostly, 0);
  }
}

void freeTraceArrs()
{
  if(cudaCalc)
  {
    hipFree(dedendx);
    hipFree(dedendz);
    hipFree(x);
    hipFree(z);
    hipFree(eden);
    hipFree(marked);
    hipFree(present);
    hipFree(boxes);
    hipFree(wpe);
    hipFree(crossesx);
    hipFree(crossesz);
  }else
  {
    delete [] dedendx;
    delete [] dedendz;
    delete [] x;
    delete [] z;
    delete [] eden;
    delete [] marked;
    delete [] present;
    delete [] boxes;
    delete [] wpe;
    delete [] crossesx;
    delete [] crossesz;
  }
}